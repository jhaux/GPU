/*********************************************************************************
 * FILENAME         alloc_and_copy.cu
 *
 * DESCRIPTION      These functions are part of the submission to exercises of 
 *                  the "GPU-Programming"
 *                  lecture of the University of Heidelberg.
 * 
 *                  Exercise 2 - Implementation of the functions used for
 *                  measureing PCIe data movements.
 *
 * AUTHORS          Christoph Klein
 *                  Marvin Kropp
 *                  Johannes Haux
 *
 * LAST CHANGE      3. Nov. 2014
 *
 ********************************************************************************/

 #include <ctime>
 #include <stdlib.h>
 #include "alloc_and_copy.h"

double allocAndCopyToHost(int sizeInBytes, bool pinned) {

    /* allocating memory on device and host for copying */
    void *dmem, *hmem;
    hipMalloc(&dmem, sizeInBytes);

    if (pinned)
        hipHostMalloc(&hmem, sizeInBytes);
    else
        void *hmem = malloc(sizeInBytes);

    /* starting timer */
    double end;
    time_t = start = time(NULL);

    /* copy data */
    hipMemcpy(hmem, dmem, sizeInBytes, hipMemcpyDeviceToHost);

    end = time(NULL);
    hipFree(dmem);
    free(hmem);

    return end - start;
}

double allocAndCopyToDevice( int sizeInBytes, bool pinned) {

    /* allocating memory on device and host for copying */
    void *dmem, *hmem;
    hipMalloc(&dmem, sizeInBytes);

    if (pinned)
        hipHostMalloc(&hmem, sizeInBytes);
    else
        void *hmem = malloc(sizeInBytes);

    /* starting timer */
    double end;
    time_t = start = time(NULL);

    /* copy data */
    hipMemcpy(dmem, hmem, sizeInBytes, hipMemcpyHostToDevice);

    end = time(NULL);
    hipFree(dmem);
    free(hmem);

    return end - start;
}
