/*********************************************************************************
 * FILENAME         main.cu
 *
 * DESCRIPTION      These functions are part of the submission to exercises of 
 *                  the "GPU-Programming"
 *                  lecture of the University of Heidelberg.
 * 
 *                  Exercise 2 - Implementation of the functions used for
 *                  measureing PCIe data movements.
 *
 * AUTHORS          Christoph Klein
 *                  Marvin Kropp
 *                  Johannes Haux
 *
 * LAST CHANGE      3. Nov. 2014
 *
 ********************************************************************************/


#include <hip/hip_runtime.h>
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>

__global__ void ongpu (double *dmem) {
//    **dmem = (double) 3.0;
    int x = 3;
    int y = 2;
    double z = 4.0;
    y = x + x + x;
    *dmem = 4.0;
}

int main() {

    time_t start = time(NULL);
    printf("%.5f\n", (double)(time(NULL) - start));
    double *dmem;
    double *hmem;
    double *hpinned;
    hmem = (double *) malloc(sizeof(double));
    hipHostMalloc(&hpinned, sizeof(double), hipHostMallocDefault);
    *hpinned = 5;
    *hmem = 3;
    std::cout << "*hpinned = " << *hpinned << std::endl;
    std::cout << "*hmem = " << *hmem << std::endl;
    hipMalloc( &dmem, sizeof(double));
    dim3 dimblock (1,1,1);
    ongpu <<< 1, dimblock >>> (dmem);

    hipMemcpy(hmem,dmem,sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(hpinned,dmem,sizeof(double),hipMemcpyDeviceToHost);
    std::cout << "*hmem = " << *hmem << std::endl;
    std::cout << "*hpinned = " << *hpinned << std::endl;
    free(hmem);
    hipFree(dmem);
//  *dmem = (double *) &x;

/*    std::cout << "dmem =" << dmem << std::endl;
    std::cout << "*dmem =" << *dmem << std::endl;
    std::cout << "**dmem =" << **dmem << std::endl;*/

    return 0;
}
