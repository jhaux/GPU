#include "hip/hip_runtime.h"
/*********************************************************************************
 * FILENAME         main.cu
 *
 * DESCRIPTION      These functions are part of the submission to exercises of 
 *                  the "GPU-Programming"
 *                  lecture of the University of Heidelberg.
 * 
 *                  Exercise 2 - Implementation of the functions used for
 *                  measureing PCIe data movements.
 *
 * AUTHORS          Christoph Klein
 *                  Marvin Kropp
 *                  Johannes Haux
 *
 * LAST CHANGE      3. Nov. 2014
 *
 ********************************************************************************/

#include <iostream> 
#include <fstream> // file output
#include <iomanip> // set precision
#include "alloc_and_copy.h"
using namespace std;

int main() {

    /*
    * Declare measurement variables
    * H2D = host to device
    */
    double *pH2D_pinned, *pH2D, *pD2H_pinned, *pD2H, *pPacketSizes;
    double startSize = 1000, endSize = 1000000;
    int N;

    pH2D_pinned =  new double[N];
    pH2D =         new double[N];
    pD2H_pinned =  new double[N];
    pD2H =         new double[N];
    pPacketSizes = new double[N];

    /* define the package sizes */
    for (int i=0; i<N; i++) {
        pPacketSizes[i] = ((double) i) * (endSize - startSize) / ((double) N) + startSize;
    }
    
    /* start measurements */
    for (int i=0; i<N; i++) {

    }

    ofstream f;
    f.open("plotdata.dat", ios::trunc); // open in replace mode
//    f << setprecision(10) 
    f.close();
    return 0;
}
